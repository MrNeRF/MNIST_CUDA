#include "hip/hip_runtime.h"
#include "error_checking.cuh"
#include "validation.cuh"
#include <vector>

__global__ void CorrectPredictionKernel(const int* predictions,
                                        const int* labels,
                                        int* d_correctCount,
                                        int batch_size);

__global__ void ArgMaxKernel(const float* predictions,
                             int* outputLabels,
                             const int numClasses,
                             const int batchSize);

float ComputeAccuracy(const int* d_predictions, const int* d_labels, int batch_size) {
    int* d_correctCount;
    hipMalloc((void**)&d_correctCount, sizeof(int));
    hipMemset(d_correctCount, 0, sizeof(int));

    CorrectPredictionKernel<<<(batch_size + 255) / 256, 256>>>(d_predictions,
                                                               d_labels,
                                                               d_correctCount,
                                                               batch_size);

    int correctCount;
    hipMemcpy(&correctCount, d_correctCount, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_correctCount);

    return (float)correctCount / batch_size;
}

void ArgMax(const float* d_predictions, int* d_labels, int numClasses, int batch_size) {

    ArgMaxKernel<<<(batch_size + 255) / 256, 256>>>(d_predictions, d_labels, 10, batch_size);
}

__global__ void CorrectPredictionKernel(const int* predictions,
                                        const int* labels,
                                        int* d_correctCount,
                                        int batch_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < batch_size) {
        if (predictions[i] == labels[i]) {
            atomicAdd(d_correctCount, 1);
        }
    }
}

__global__ void ArgMaxKernel(const float* predictions,
                             int* outputLabels,
                             const int num_classes,
                             const int batch_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= batch_size)
        return;

    int predictedClass = 0;
    float maxProb = predictions[idx * num_classes];

    for (int i = 1; i < num_classes; ++i) {
        float prob = predictions[idx * num_classes + i];
        if (prob > maxProb) {
            predictedClass = i;
            maxProb = prob;
        }
    }

    outputLabels[idx] = predictedClass;
}